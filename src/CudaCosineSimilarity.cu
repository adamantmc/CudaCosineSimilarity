#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include "CudaCosineSimilarity.hpp"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

// ==========
// Macro taken from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// ==========

__device__ int getElementsPerUnit(int total, int number_of_units) {
    int elements_per_unit = total / number_of_units;
    double remains = total % number_of_units;

    if(remains != 0) {
        elements_per_unit += 1;
    }

    return elements_per_unit;
}

__device__ double dotProduct(double *a, double *b, int size) {
    double result = 0;

    for(int i = 0; i < size; i++) {
        result += a[i] * b[i];
    }

    return result;
}

__global__ void dotProductKernel(double *a, double *b, int a_size, int b_size, double *results, int vector_size) {
    int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
    int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

    // Get range of 'a' vectors we will work with
    int a_start = blockIdx.x * a_vectors_per_block;
    int a_end = a_start + a_vectors_per_block;

    if(a_end > a_size) {
        a_end = a_size;
    }

    // Get range of 'b' vectors we will work with
    int b_start = threadIdx.x * b_vectors_per_thread;
    int b_end = b_start + b_vectors_per_thread;

    if(b_end > b_size) {
        b_end = b_size;
    }

    if(a_start < a_size && b_start < b_size) {
        for(int a_index = a_start; a_index < a_end; a_index++) {
            for(int b_index = b_start; b_index < b_end; b_index++) {
                results[a_index*b_size + b_index] = dotProduct(&a[a_index*vector_size], &b[b_index*vector_size], vector_size);
            }
        }
    }
}

__global__ void normKernel(double *vectors, int size, double *results, int vector_size) {
    int vectors_per_block = getElementsPerUnit(size, gridDim.x);

    // Get range of vectors we will work with
    int start = blockIdx.x * vectors_per_block;
    int end = start + vectors_per_block;

    if(end > size) {
        end = size;
    }

    for(int vec_index = start; vec_index < end; vec_index++) {
        for(int i = 0; i < vector_size; i++) {
            results[vec_index] += pow(vectors[vec_index*vector_size + i], 2);
        }

        results[vec_index] = sqrt(results[vec_index]);
    }
}

__global__ void cosineSimilarityKernel(double *dot_products, int a_size, int b_size, double *a_norms, double *b_norms, double *results) {
    int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
    int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

    int a_start = blockIdx.x * a_vectors_per_block;
    int a_end = a_start + a_vectors_per_block;

    if(a_end > a_size) {
        a_end = a_size;
    }

    int b_start = threadIdx.x * b_vectors_per_thread;
    int b_end = b_start + b_vectors_per_thread;

    if(b_end > b_size) {
        b_end = b_size;
    }

    for(int a_index = a_start; a_index < a_end; a_index++) {
        for(int b_index = b_start; b_index < b_end; b_index++) {
            results[a_index*b_size + b_index] = (double) dot_products[a_index*b_size + b_index] / (a_norms[a_index] * b_norms[b_index]);
        }
    }
}

double *cudaCosine(std::vector<std::vector<double> > v1, std::vector<std::vector<double> > v2) {
    // Get size of first vector of v1 (same for all vectors, v2 also)
    int vector_size = v1[0].size();

    std::cout << "V1 Size: " << v1.size() << " V2 Size: " << v2.size();
    std::cout << " V1 Vector Size: " << v1[0].size() << " V2 Vector Size: " << v2[0].size();
    std::cout << " Results Size: " << v1.size()*v2.size() << std::endl;

    int v1_total_length = v1.size()*vector_size;
    int v2_total_length = v2.size()*vector_size;

    double *v1_ptr = new double[v1_total_length];
    double *v2_ptr = new double[v2_total_length];
    double *results = new double[v1.size()*v2.size()];

    for(int i = 0; i < v1.size(); i++) {
        for(int j = 0; j < vector_size; j++) {
            v1_ptr[i*vector_size + j] = v1[i][j];
        }
    }

    for(int i = 0; i < v2.size(); i++) {
        for(int j = 0; j < vector_size; j++) {
            v2_ptr[i*vector_size + j] = v2[i][j];
        }
    }

    size_t free_mem, total_mem;
    gpuErrchk(hipMemGetInfo(&free_mem, &total_mem));
    std::cout << "GPU Memory: " << free_mem/1000 << " KB free, " << total_mem/1000 << " KB total." << std::endl;

    size_t v1_size = v1.size() * vector_size * sizeof(double);
    size_t v2_size = v2.size() * vector_size * sizeof(double);
    size_t v1_norms_size = v1.size() * sizeof(double);
    size_t v2_norms_size = v2.size() * sizeof(double);
    size_t results_size = v1.size() * v2.size() * sizeof(double);

    double *cu_v1_ptr, *cu_v2_ptr;
    double *cu_v1_norms, *cu_v2_norms;
    double *cu_dot_products;

    std::cout << "Allocating memory in the GPU" << std::endl;
    std::cout << "Allocating " << v1_size/1000 << " KB for V1" << std::endl;
    gpuErrchk(hipMalloc((void **) &cu_v1_ptr, v1_size));
    std::cout << "Allocating " << v2_size/1000 << " KB for V2" << std::endl;
    gpuErrchk(hipMalloc((void **) &cu_v2_ptr, v2_size));

    std::cout << "Allocating " << results_size/1000 << " KB for Dot Products" << std::endl;
    gpuErrchk(hipMalloc((void **) &cu_dot_products, results_size));
    std::cout << "Allocating " << v1_norms_size/1000 << " KB for V1 Norms" << std::endl;
    gpuErrchk(hipMalloc((void **) &cu_v1_norms, v1_norms_size));
    std::cout << "Allocating " << v2_norms_size/1000 << " KB for V2 Norms" << std::endl;
    gpuErrchk(hipMalloc((void **) &cu_v2_norms, v2_norms_size));
    std::cout << "Initializing Dot Products array with zero values" << std::endl;
    gpuErrchk(hipMemset(cu_dot_products, 0, results_size));

    std::cout << "Copying V1 vectors to the GPU" << std::endl;
    gpuErrchk(hipMemcpy(cu_v1_ptr, v1_ptr, v1_size, hipMemcpyHostToDevice));
    std::cout << "Copying V2 vectors to the GPU" << std::endl;
    gpuErrchk(hipMemcpy(cu_v2_ptr, v2_ptr, v2_size, hipMemcpyHostToDevice));

    hipDeviceSynchronize();

    std::cout << "Executing Dot-Product Kernel on " << NUMBER_OF_BLOCKS << " Blocks and " << NUMBER_OF_THREADS << " Threads." << std::endl;
    dotProductKernel<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS>>>(cu_v1_ptr, cu_v2_ptr, v1.size(), v2.size(), cu_dot_products, vector_size);

    std::cout << "Executing V1 Norms Kernel" << std::endl;
    normKernel<<<NUMBER_OF_BLOCKS, 1>>>(cu_v1_ptr, v1.size(), cu_v1_norms, vector_size);
    std::cout << "Executing V2 Norms Kernel" << std::endl;
    normKernel<<<NUMBER_OF_BLOCKS, 1>>>(cu_v2_ptr, v2.size(), cu_v2_norms, vector_size);

    hipDeviceSynchronize();

    std::cout << "Executing Cosine Similarity Kernel" << std::endl;
    cosineSimilarityKernel<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS>>>(cu_dot_products, v1.size(), v2.size(), cu_v1_norms, cu_v2_norms, cu_dot_products);

    hipDeviceSynchronize();

    std::cout << "Getting results from GPU memory" << std::endl;
    gpuErrchk(hipMemcpy(results, cu_dot_products, results_size, hipMemcpyDeviceToHost));

    hipFree(cu_v1_ptr);
    hipFree(cu_v2_ptr);
    hipFree(cu_v1_norms);
    hipFree(cu_v2_norms);
    hipFree(cu_dot_products);

    delete v1_ptr;
    delete v2_ptr;

    return results;
}
